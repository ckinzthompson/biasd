#include <stdlib.h>
#include <hip/hip_runtime.h>

int device_count() {
	int count;
	hipGetDeviceCount(&count);
	return count;
}

int cuda_errors(int device){
	hipSetDevice(device);
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("CUDA Error : %s\n", hipGetErrorString(err));
		return 1;
	}
	return 0;
}
